//Darrien Park

#include "hip/hip_runtime.h"
#include <string.h>
#include <stdio.h>

//no field in cudaDeviceProperties for number of cores. Therefore need to determine based on compute capability
int getCores(hipDeviceProp_t dev_prop)
{
	int cores = 0;
	int sm = dev_prop.multiProcessorCount;

	//start switch case based on major compute capability
	switch (dev_prop.major){
		//Fermi
		case 2:
			if (dev_prop.minor == 1)
				cores = sm * 48;
			else cores = sm * 32;
				break;
		//Kepler
		case 3:
			cores = sm * 192;
				break;
		//Maxwell
		case 5:
			cores = sm * 128;
				break;
		//Pascal
		case 6:
			if (dev_prop.minor == 1)
				cores = sm * 128;
			else if (dev_prop.minor == 0)
				cores = sm * 64;
			else printf("Unknown device type \n");
				break;
		//Volta
		case 7:
			if (dev_prop.minor == 0)
				cores = sm * 64;
			else printf("Unknown device type \n");
				break;
		//base case: can't be detected
		default:
			printf("Unknown device type \n");
				break;
	}
	return cores;
}

int main(int argc, char * argv[])
{

	int dev_count;
	hipGetDeviceCount(& dev_count);
	printf("Number of CUDA devices is [%d]\n\n",dev_count);

	for(int i = 0; i < dev_count; i++){
		int k = i+1;
		printf("Device [%d]\n", k);

		hipDeviceProp_t dev_props;

		hipGetDeviceProperties(&dev_props, 0);						//hipGetDeviceProperties(hipDeviceProp_t* prop, int device#)
		printf("	Device Name: %s\n",dev_props.name);
		printf("	Memory Clock Rate (KHz): %d\n",dev_props.memoryClockRate);
		printf("	Number of Streaming Multiprocessors: %d\n",dev_props.multiProcessorCount);
		printf("	Number of cores: %d\n",getCores(dev_props));
		printf("	Warp Size: %d\n",dev_props.warpSize);
		printf("	Total Global Memory: %d\n",dev_props.totalGlobalMem);
		printf("	Total Constant Memory: %d\n",dev_props.totalConstMem);
		printf("	Shared Memory/Block: %d\n",dev_props.sharedMemPerBlock);
		printf("	Number of Registers/Block: %d\n",dev_props.regsPerBlock);
		printf("	Number of Threads/Block: %d\n",dev_props.maxThreadsPerBlock);
		printf("	Max Block Dimension: %d\n",dev_props.maxThreadsDim);
		printf("	Max Grid Dimension: %d\n",dev_props.maxGridSize);
	}
	return 0;
}
